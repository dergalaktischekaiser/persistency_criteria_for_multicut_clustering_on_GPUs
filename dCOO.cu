#include "hip/hip_runtime.h"
/*source: https://github.com/pawelswoboda/RAMA */
#include "dCOO.h"
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include "time_measure_util.h"
#include "rama_utils.h"


__global__ void map_nodes(const int num_edges, const int* const __restrict__ node_mapping, int* __restrict__ rows, int* __restrict__ cols)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int e = tid; e < num_edges; e += num_threads)
    {
        assert(rows[e] < cols[e]);
        const int i = node_mapping[rows[e]];
        const int j = node_mapping[cols[e]];
        rows[e] = min(i, j);
        cols[e] = max(i, j);
    }
}

void dCOO::init(const bool is_sorted)
{
    assert(col_ids.size() == data.size());
    assert(row_ids.size() == data.size());
    std::cout << "\nassertion 1 passed\n";
    if(is_sorted)
    {
        assert(thrust::is_sorted(row_ids.begin(), row_ids.end())); 
        assert(thrust::is_sorted(thrust::make_zip_iterator(thrust::make_tuple(row_ids.begin(), col_ids.begin())),
                                thrust::make_zip_iterator(thrust::make_tuple(row_ids.end(), col_ids.end())))); 
    }
    else
    {
        if (is_directed_)
            sort_edge_nodes(row_ids, col_ids);
        coo_sorting(row_ids, col_ids, data);
        // now row indices are non-decreasing
        assert(thrust::is_sorted(row_ids.begin(), row_ids.end()));
        std::cout << "assertion 2 passed\n";
    } 

    if(cols_ == 0)
        cols_ = *thrust::max_element(col_ids.begin(), col_ids.end()) + 1;
    assert(cols_ > *thrust::max_element(col_ids.begin(), col_ids.end()));
    if(rows_ == 0)
        rows_ = row_ids.back() + 1;
    assert(rows_ > *thrust::max_element(row_ids.begin(), row_ids.end()));
    if (!is_directed_)
        assert(rows_ == cols_);
    std::cout << "all assertions passed\n";
}

dCOO dCOO::contract_cuda(const thrust::device_vector<int>& node_mapping)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    assert(is_directed_);

    const int numThreads = 256;

    thrust::device_vector<int> new_row_ids = row_ids;
    thrust::device_vector<int> new_col_ids = col_ids;
    thrust::device_vector<float> new_data = data;

    int num_edges = new_row_ids.size();
    int numBlocks = ceil(num_edges / (float) numThreads);
    map_nodes<<<numBlocks, numThreads>>>(num_edges, 
            thrust::raw_pointer_cast(node_mapping.data()), 
            thrust::raw_pointer_cast(new_row_ids.data()), 
            thrust::raw_pointer_cast(new_col_ids.data()));

    coo_sorting(new_row_ids, new_col_ids, new_data); // in-place sorting by rows.

    auto first = thrust::make_zip_iterator(thrust::make_tuple(new_row_ids.begin(), new_col_ids.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(new_row_ids.end(), new_col_ids.end()));

    thrust::device_vector<int> out_rows(num_edges);
    thrust::device_vector<int> out_cols(num_edges);
    auto first_output = thrust::make_zip_iterator(thrust::make_tuple(out_rows.begin(), out_cols.begin()));
    thrust::device_vector<float> out_data(num_edges);

    auto new_end = thrust::reduce_by_key(first, last, new_data.begin(), first_output, out_data.begin());
    int new_num_edges = std::distance(out_data.begin(), new_end.second);
    out_rows.resize(new_num_edges);
    out_cols.resize(new_num_edges);
    out_data.resize(new_num_edges);

    int out_num_rows = out_rows.back() + 1;
    int out_num_cols = *thrust::max_element(out_cols.begin(), out_cols.end()) + 1;

    return dCOO(out_num_rows, out_num_cols, std::move(out_cols), std::move(out_rows), std::move(out_data), is_directed_, true);
}

struct is_diagonal
{
    __host__ __device__
        bool operator()(thrust::tuple<int,int,float> t)
        {
            return thrust::get<0>(t) == thrust::get<1>(t);
        }
};

void dCOO::remove_diagonal()
{
    auto begin = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), row_ids.begin(), data.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), row_ids.end(), data.end()));

    auto new_last = thrust::remove_if(begin, end, is_diagonal());
    int new_num_edges = std::distance(begin, new_last);
    col_ids.resize(new_num_edges);
    row_ids.resize(new_num_edges);
    data.resize(new_num_edges);
}

struct diag_func
{
    float* d;
    __host__ __device__
        void operator()(thrust::tuple<int,int,float> t)
        {
            if(thrust::get<0>(t) == thrust::get<1>(t))
            {
                assert(d[thrust::get<0>(t)] == 0.0);
                d[thrust::get<0>(t)] = thrust::get<2>(t);
            }
        }
};

thrust::device_vector<float> dCOO::diagonal() const
{
    thrust::device_vector<float> d(std::max(rows(), cols()), 0.0);

    auto begin = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), row_ids.begin(), data.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), row_ids.end(), data.end()));

    thrust::for_each(begin, end, diag_func({thrust::raw_pointer_cast(d.data())})); 

    return d;
}

thrust::device_vector<int> dCOO::compute_row_offsets() const
{
    return compute_offsets(row_ids, rows_ - 1);
}

float dCOO::sum() const
{
    return thrust::reduce(data.begin(), data.end(), (float) 0.0, thrust::plus<float>());
}

float dCOO::min() const
{
    return *thrust::min_element(data.begin(), data.end()); 
}

float dCOO::max() const
{
    return *thrust::max_element(data.begin(), data.end()); 
}

dCOO dCOO::export_undirected() const
{
    assert(is_directed_);
    thrust::device_vector<int> row_ids_u, col_ids_u;
    thrust::device_vector<float> data_u;

    std::tie(row_ids_u, col_ids_u, data_u) = to_undirected(row_ids, col_ids, data);
    return dCOO(std::move(col_ids_u), std::move(row_ids_u), std::move(data_u), false);
}

dCOO dCOO::export_directed() const
{
    assert(!is_directed_);
    thrust::device_vector<int> row_ids_d, col_ids_d;
    thrust::device_vector<float> data_d;

    std::tie(row_ids_d, col_ids_d, data_d) = to_directed(row_ids, col_ids, data);
    return dCOO(std::move(col_ids_d), std::move(row_ids_d), std::move(data_d), true);
}

struct is_in_range
{
    const float lb;
    const float ub;

      __host__ __device__
            bool operator()(const float x)
            {
                if(x >= lb && x <= ub)
                    return true;
                else
                    return false;
            }

      __host__ __device__
            bool operator()(const thrust::tuple<int,int,float> t)
            {
                return operator()(thrust::get<2>(t));
            }
};

dCOO dCOO::export_filtered(const float lb, const float ub) const
{
    assert(lb <= ub);
    const int new_nnz = thrust::count_if(data.begin(), data.end(), is_in_range({lb,ub}));
    thrust::device_vector<int> col_ids_f(new_nnz), row_ids_f(new_nnz);
    thrust::device_vector<float> data_f(new_nnz);

    auto first = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), row_ids.begin(), data.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), row_ids.end(), data.end()));

    auto first_f = thrust::make_zip_iterator(thrust::make_tuple(col_ids_f.begin(), row_ids_f.begin(), data_f.begin()));

    thrust::copy_if(first, last, first_f, is_in_range({lb,ub}));

    return dCOO(rows(), cols(), 
            std::move(col_ids_f), std::move(row_ids_f), std::move(data_f), is_directed_, true); 
}

void dCOO::print() const
{
    std::cout<<"A: \n";
    print_vector(row_ids, "row_ids");
    print_vector(col_ids, "col_ids");
    print_vector(data   , "data   ");
    std::cout<<"\n";
}